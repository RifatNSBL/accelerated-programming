
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  int NUM_THREAD = 256;
  int NUM_BLOCK = 32;
  hipError_t syncErr, asyncErr;

  addVectorsInto<<<NUM_BLOCK, NUM_THREAD>>>(c, a, b, N);
  
  syncErr = hipGetLastError();
  asyncErr = hipDeviceSynchronize();

  if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
  
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
